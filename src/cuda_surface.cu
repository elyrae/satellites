#include "hip/hip_runtime.h"
#include "cuda_surface.cuh"

#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void CUDA_Surface::Points3D::allocate(const int _size)
{
    size = _size;
    hipMalloc((void**) &x, size*sizeof(float));
    hipMalloc((void**) &y, size*sizeof(float));
    hipMalloc((void**) &z, size*sizeof(float));
}

void CUDA_Surface::Points3D::free()
{
    hipFree(x);
    hipFree(y);
    hipFree(z);
}

void CUDA_Surface::Points3D::load_from(const CUDA_Surface::Points3D &cpu) const
{
    hipMemcpy(x, cpu.x, size*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(y, cpu.y, size*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(z, cpu.z, size*sizeof(float), hipMemcpyHostToDevice);
}

// void CUDA_Surface::Points3D::save_to(const CUDA_Surface::Points3D &cpu) const
// {
//     hipMemcpy(cpu.x, x, size*sizeof(float), hipMemcpyDeviceToHost);
//     hipMemcpy(cpu.y, y, size*sizeof(float), hipMemcpyDeviceToHost);
//     hipMemcpy(cpu.z, z, size*sizeof(float), hipMemcpyDeviceToHost);
// }

void CUDA_Surface::Points1D::allocate(const int _size)
{
    size = _size;
    hipMalloc((void**) &s, size*sizeof(int));
    hipMemset(s, 0, size);
}

void CUDA_Surface::Points1D::free()
{
    hipFree(s);
}

// void CUDA_Surface::Points1D::load_from(const CUDA_Surface::Points1D &cpu) const
// {
//     hipMemcpy(s, cpu.s, size*sizeof(int), hipMemcpyHostToDevice);
// }

void CUDA_Surface::Points1D::save_to(const CUDA_Surface::Points1D &cpu) const
{
    hipMemcpy(cpu.s, s, size*sizeof(int), hipMemcpyDeviceToHost);
}

__global__ void compute_surface_layers(float *surface, 
    const float *cx, const float *cy, const float *cz, 
    const float *x,  const float *y,  const float *z, const float hor)
{
    const int SURFACE_SIZE       = 5120;
    const int CONFIGURATION_SIZE = 20;
    const int CONFIGURATIONS     = 256;

    const int global_conf  = blockDim.x*blockIdx.x + threadIdx.x; // configuration 
    const int global_point = blockDim.y*blockIdx.y + threadIdx.y; // point
    const int global_time  = blockDim.z*blockIdx.z + threadIdx.z; // time

    const float cen_x = cx[global_point];
    const float cen_y = cy[global_point];
    const float cen_z = cz[global_point];

    float surface_result = 0.0; 
    float sat_x = 0.0, sat_y = 0.0, sat_z = 0.0; 
    for (int iorb = 0; iorb < CONFIGURATION_SIZE; iorb++) 
    {
        sat_x = x[(global_time*CONFIGURATION_SIZE + iorb)*CONFIGURATIONS + global_conf];
        sat_y = y[(global_time*CONFIGURATION_SIZE + iorb)*CONFIGURATIONS + global_conf];
        sat_z = z[(global_time*CONFIGURATION_SIZE + iorb)*CONFIGURATIONS + global_conf];
        surface_result += (sat_x*cen_x + sat_y*cen_y + sat_z*cen_z > hor);
    }
    surface[(global_time*SURFACE_SIZE + global_point)*CONFIGURATIONS + global_conf] = surface_result;
}

__global__ void reduce_time(float *surface, float *max_time)
{
    const int SURFACE_SIZE       = 5120;
    const int CONFIGURATIONS     = 256;
    const int TIMESTEPS          = 240;

    const int global_conf  = blockDim.x*blockIdx.x + threadIdx.x; // configuration 
    const int global_point = blockDim.y*blockIdx.y + threadIdx.y; // point

    float surf = 0.0; 
    float time = 0.0, m = 0.0; 
    for (int timestep = 1; timestep < TIMESTEPS; timestep++) 
    {
        surf = surface[(timestep*SURFACE_SIZE + global_point)*CONFIGURATIONS + global_conf];
        m    = (surf == 0.0) ? fmaxf(m, time) :       m;
        time = (surf == 0.0) ? 0.0            : (time + 30.0);  
    }
    m = fmaxf(m, time);
    max_time[global_point*CONFIGURATIONS + global_conf] = m;
}

// __global__ void reduce_time(float *surface, float *max_time)
// {
//     const int SURFACE_SIZE       = 5120;
//     const int CONFIGURATIONS     = 256;
//     const int TIMESTEPS          = 240;

//     const int global_conf  = blockDim.x*blockIdx.x + threadIdx.x; // configuration 
//     const int global_point = blockDim.y*blockIdx.y + threadIdx.y; // point

//     float surf = 0.0; 
//     float time = 0.0, m = 0.0, surface_result = 0.0; 
//     for (int timestep = 1; timestep < TIMESTEPS; timestep++) {
//         for (int iorb = 0; iorb < CONFIGURATION_SIZE; iorb++)
//         {
//             sat_x = x[(global_time*CONFIGURATION_SIZE + iorb)*CONFIGURATIONS + global_conf];
//             sat_y = y[(global_time*CONFIGURATION_SIZE + iorb)*CONFIGURATIONS + global_conf];
//             sat_z = z[(global_time*CONFIGURATION_SIZE + iorb)*CONFIGURATIONS + global_conf];
//             surface_result += (sat_x*cen_x + sat_y*cen_y + sat_z*cen_z > hor);
//         }
//         m    = (surf == 0.0) ? fmaxf(m, time) :       m;
//         time = (surf == 0.0) ? 0.0            : (time + 30.0);  
//     }
//     m = fmaxf(m, time);
//     max_time[global_point*CONFIGURATIONS + global_conf] = m;
// }

__global__ void reduce_points(float *max_time)
{
    const int SURFACE_SIZE   = 5120;
    const int CONFIGURATIONS = 256;

    const int global_conf  = blockDim.x*blockIdx.x + threadIdx.x; // configuration 

    float max = 0.0; 
    for (int point = 0; point < SURFACE_SIZE; point++)
        max = fmaxf(max, max_time[point*CONFIGURATIONS + global_conf]);
    max_time[global_conf] = max;
}

float loc_horizon(const float H, const float alpha)
{
    const float delta = 10.0 * M_PI / 180.0; // требуемое возвышение спутника над горизонтом
    const float alpha_star = asin(cos(delta) / H);

    return (alpha < alpha_star) ? cos(asin(H*sin(alpha)) - alpha) : sin(delta + alpha_star);
    // return sin(delta + alpha_star);
}

float CUDA_Surface::compute_surface(Points1D &gpu_surface, Points1D &gpu_max_time, 
                                    const Points3D &gpu_centroids, const Points3D &gpu_pos)
{
    const int SURFACE_SIZE       = 5120;
    const int CONFIGURATION_SIZE = 20;
    const int CONFIGURATIONS     = 256;
    const int TIMESTEPS = 240;

    dim3    dim_grid_surf(1, SURFACE_SIZE, TIMESTEPS),    dim_block_surf(CONFIGURATIONS, 1, 1);
    dim3 dim_grid_maxtime(1, SURFACE_SIZE, 1),         dim_block_maxtime(CONFIGURATIONS, 1, 1);
    dim3  dim_grid_points(1, 1,            1),          dim_block_points(CONFIGURATIONS, 1, 1);

    const float H = (6371.0 + 1500.0) / 6371.0;
    const float alpha = (120.0 * M_PI / 180.0) / 2.0;
    const float hor = loc_horizon(H, alpha);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    compute_surface_layers<<<dim_grid_surf, dim_block_surf>>>(gpu_surface.s, 
                                                              gpu_centroids.x, gpu_centroids.y, gpu_centroids.z,
                                                              gpu_pos.x,       gpu_pos.y,       gpu_pos.z, hor);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float gpu_time = 0.0; 
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("surf = %f ms\n", gpu_time);

    hipEventRecord(start, 0);
    reduce_time<<<dim_grid_maxtime, dim_block_maxtime>>>(gpu_surface.s, gpu_max_time.s);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&gpu_time, start, stop);
    printf("time reduce = %f ms\n", gpu_time);
    
    hipEventRecord(start, 0);
    reduce_points<<<dim_grid_points, dim_block_points>>>(gpu_max_time.s);
    // gpuErrchk( hipPeekAtLastError() );
    // gpuErrchk( hipDeviceSynchronize() );    

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&gpu_time, start, stop);
    printf("point reduce = %f ms\n", gpu_time);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    return gpu_time;          
} 
