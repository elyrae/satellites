#include "hip/hip_runtime.h"
#include "cuda_surface.cuh"

#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>

void CUDA_Surface::Points3D::allocate()
{
    hipMalloc((void**) &x, size*sizeof(float));
    hipMalloc((void**) &y, size*sizeof(float));
    hipMalloc((void**) &z, size*sizeof(float));
}

void CUDA_Surface::Points3D::allocate(const int _size)
{
    size = _size;
    hipMalloc((void**) &x, size*sizeof(float));
    hipMalloc((void**) &y, size*sizeof(float));
    hipMalloc((void**) &z, size*sizeof(float));
}

void CUDA_Surface::Points3D::free()
{
    hipFree(x);
    hipFree(y);
    hipFree(z);
}

void CUDA_Surface::Points3D::load_from(const CUDA_Surface::Points3D &cpu) const
{
    hipMemcpy(x, cpu.x, size*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(y, cpu.y, size*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(z, cpu.z, size*sizeof(float), hipMemcpyHostToDevice);
}

void CUDA_Surface::Points3D::save_to(const CUDA_Surface::Points3D &cpu) const
{
    hipMemcpy(cpu.x, x, size*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(cpu.y, y, size*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(cpu.z, z, size*sizeof(float), hipMemcpyDeviceToHost);
}

void CUDA_Surface::Points1D::allocate()
{
    hipMalloc((void**) &s, size*sizeof(float));
    hipMemset(s, 0, size);
}

void CUDA_Surface::Points1D::allocate(const int _size)
{
    size = _size;
    hipMalloc((void**) &s, size*sizeof(float));
    hipMemset(s, 0, size);
}

void CUDA_Surface::Points1D::free()
{
    hipFree(s);
}

void CUDA_Surface::Points1D::load_from(const CUDA_Surface::Points1D &cpu) const
{
    hipMemcpy(s, cpu.s, size*sizeof(float), hipMemcpyHostToDevice);
}

void CUDA_Surface::Points1D::save_to(const CUDA_Surface::Points1D &cpu) const
{
    hipMemcpy(cpu.s, s, size*sizeof(float), hipMemcpyDeviceToHost);
}


// __global__ void compute_surface_layer(float *surface, const float *cx, const float *cy, const float *cz, 
//                                                       const float *x, const float *y, const float *z, const float hor)
// {
//     const int SURFACE_SIZE       = 5120;
//     const int CONFIGURATION_SIZE = 20;
//     const int CONFIGURATIONS     = 100;
//     const int TIMESTEPS          = 240;

//     const int global_conf  = blockDim.x*blockIdx.x + threadIdx.x; // configuration 
//     const int global_point = blockDim.y*blockIdx.y + threadIdx.y; // point
//     const int global_time  = blockDim.z*blockIdx.z + threadIdx.z; // time

//     float surface_result = 0.0, sat_x = 0.0, sat_y = 0.0, sat_z = 0.0; 
//     for (int iorb = 0; iorb < CONFIGURATION_SIZE; iorb++)
//     {
//         sat_x = x[(global_time*CONFIGURATIONS + global_conf)*CONFIGURATION_SIZE + iorb];
//         sat_y = y[(global_time*CONFIGURATIONS + global_conf)*CONFIGURATION_SIZE + iorb];
//         sat_z = z[(global_time*CONFIGURATIONS + global_conf)*CONFIGURATION_SIZE + iorb];
//         // sat_h = h[(global_time*CONFIGURATIONS + global_conf)*CONFIGURATION_SIZE + iorb]    
//         surface_result |= (sat_x*cx[global_point] + sat_y*cy[global_point] + sat_z*cz[global_point] > hor);
//     } 
//     surface[(global_time*SURFACE_SIZE + global_point)*CONFIGURATIONS + global_conf] = surface_result;
// }

__global__ void compute_surface_layer(float *surface, 
    const float *cx, const float *cy, const float *cz, 
    const float *x,  const float *y,  const float *z, const float hor)
{
    const int SURFACE_SIZE       = 5120;
    const int CONFIGURATION_SIZE = 20;
    const int CONFIGURATIONS     = 128;

    const int global_conf  = blockDim.x*blockIdx.x + threadIdx.x; // configuration 
    const int global_point = blockDim.y*blockIdx.y + threadIdx.y; // point
    const int global_time  = blockDim.z*blockIdx.z + threadIdx.z; // time

    float surface_result = 0.0, sat_x = 0.0, sat_y = 0.0, sat_z = 0.0; 
    for (int iorb = 0; iorb < CONFIGURATION_SIZE; iorb++)
    {
        sat_x = x[(global_time*CONFIGURATION_SIZE + iorb)*CONFIGURATIONS + global_conf];
        sat_y = y[(global_time*CONFIGURATION_SIZE + iorb)*CONFIGURATIONS + global_conf];
        sat_z = z[(global_time*CONFIGURATION_SIZE + iorb)*CONFIGURATIONS + global_conf];
        surface_result += (sat_x*cx[global_point] + sat_y*cy[global_point] + sat_z*cz[global_point] > hor);
    } 
    surface[(global_time*SURFACE_SIZE + global_point)*CONFIGURATIONS + global_conf] = surface_result;
}

float loc_horizon(const float H, const float alpha)
{
    const float delta = 10.0 * M_PI / 180.0; // требуемое возвышение спутника над горизонтом
    const float alpha_star = asin(cos(delta) / H);

    return (alpha < alpha_star) ? cos(asin(H*sin(alpha)) - alpha) : sin(delta + alpha_star);
    // return sin(delta + alpha_star);
}

float CUDA_Surface::compute_surface(Points1D &gpu_surface, const Points3D &gpu_centroids, const Points3D &gpu_pos)
{
    const int SURFACE_SIZE       = 5120;
    // const int CONFIGURATION_SIZE = 20;
    const int CONFIGURATIONS     = 128;
    const int TIMESTEPS = 240;

    dim3 dim_grid, dim_block;

    dim_block.x = CONFIGURATIONS;
    dim_block.y = 1;
    dim_block.z = 1;

    dim_grid.x = 1;
    dim_grid.y = SURFACE_SIZE;
    dim_grid.z = TIMESTEPS;

    const float H = (6371.0 + 1500.0) / 6371.0;
    const float alpha = (120.0 * M_PI / 180.0) / 2.0;
    const float hor = loc_horizon(H, alpha);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    compute_surface_layer<<<dim_grid, dim_block>>>(gpu_surface.s, gpu_centroids.x, gpu_centroids.y, gpu_centroids.z,
                                                                  gpu_pos.x,       gpu_pos.y,       gpu_pos.z, hor);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float gpu_time = 0.0; 
    hipEventElapsedTime(&gpu_time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return gpu_time;          
} 