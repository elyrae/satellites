#include "hip/hip_runtime.h"
#include "cuda_surface.cuh"

#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void CUDA_Surface::Points3D::allocate(const int _size)
{
    size = _size;
    hipMalloc((void**) &x, size*sizeof(float));
    hipMalloc((void**) &y, size*sizeof(float));
    hipMalloc((void**) &z, size*sizeof(float));
}

void CUDA_Surface::Points3D::free()
{
    hipFree(x);
    hipFree(y);
    hipFree(z);
}

void CUDA_Surface::Points3D::load_from(const CUDA_Surface::Points3D &cpu) const
{
    hipMemcpy(x, cpu.x, size*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(y, cpu.y, size*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(z, cpu.z, size*sizeof(float), hipMemcpyHostToDevice);
}

void CUDA_Surface::Points3D::save_to(const CUDA_Surface::Points3D &cpu) const
{
    hipMemcpy(cpu.x, x, size*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(cpu.y, y, size*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(cpu.z, z, size*sizeof(float), hipMemcpyDeviceToHost);
}

void CUDA_Surface::Points1D::allocate(const int _size)
{
    size = _size;
    hipMalloc((void**) &s, size*sizeof(int));
    hipMemset(s, 0, size);
}

void CUDA_Surface::Points1D::free()
{
    hipFree(s);
}

void CUDA_Surface::Points1D::load_from(const CUDA_Surface::Points1D &cpu) const
{
    hipMemcpy(s, cpu.s, size*sizeof(int), hipMemcpyHostToDevice);
}

void CUDA_Surface::Points1D::save_to(const CUDA_Surface::Points1D &cpu) const
{
    hipMemcpy(cpu.s, s, size*sizeof(int), hipMemcpyDeviceToHost);
}


// __global__ void compute_surface_layer(float *surface, const float4 *c, const float4 *p, const float hor)
// {
//     const int SURFACE_SIZE       = 5120;
//     const int CONFIGURATION_SIZE = 20;
//     const int CONFIGURATIONS     = 128;

//     const int global_conf  = blockDim.x*blockIdx.x + threadIdx.x; // configuration 
//     const int global_point = blockDim.y*blockIdx.y + threadIdx.y; // point
//     const int global_time  = blockDim.z*blockIdx.z + threadIdx.z; // time

//     float surface_result = 0.0, sat_x = 0.0, sat_y = 0.0, sat_z = 0.0;
//     const float cen_x = c[global_point].x;
//     const float cen_y = c[global_point].y;
//     const float cen_z = c[global_point].z;

//     for (int iorb = 0; iorb < CONFIGURATION_SIZE; iorb++)
//     {
//         sat_x = p[(global_time*CONFIGURATION_SIZE + iorb)*CONFIGURATIONS + global_conf].x;
//         sat_y = p[(global_time*CONFIGURATION_SIZE + iorb)*CONFIGURATIONS + global_conf].y;
//         sat_z = p[(global_time*CONFIGURATION_SIZE + iorb)*CONFIGURATIONS + global_conf].z;
//         surface_result += (sat_x*cen_x + sat_y*cen_y + sat_z*cen_z > hor);
//     } 
//     surface[(global_time*SURFACE_SIZE + global_point)*CONFIGURATIONS + global_conf] = surface_result;
// }

__global__ void compute_surface_layers(float *surface, 
    const float *cx, const float *cy, const float *cz, 
    const float *x,  const float *y,  const float *z, const float hor)
{
    const int SURFACE_SIZE       = 5120;
    const int CONFIGURATION_SIZE = 20;
    const int CONFIGURATIONS     = 256;

    const int global_conf  = blockDim.x*blockIdx.x + threadIdx.x; // configuration 
    const int global_point = blockDim.y*blockIdx.y + threadIdx.y; // point
    const int global_time  = blockDim.z*blockIdx.z + threadIdx.z; // time

    const float cen_x = cx[global_point];
    const float cen_y = cy[global_point];
    const float cen_z = cz[global_point];

    float surface_result = 0.0; 
    float sat_x = 0.0, sat_y = 0.0, sat_z = 0.0; 
    for (int iorb = 0; iorb < CONFIGURATION_SIZE; iorb++)
    {
        sat_x = x[(global_time*CONFIGURATION_SIZE + iorb)*CONFIGURATIONS + global_conf];
        sat_y = y[(global_time*CONFIGURATION_SIZE + iorb)*CONFIGURATIONS + global_conf];
        sat_z = z[(global_time*CONFIGURATION_SIZE + iorb)*CONFIGURATIONS + global_conf];
        surface_result += (sat_x*cen_x + sat_y*cen_y + sat_z*cen_z > hor);
    } 
    surface[(global_time*SURFACE_SIZE + global_point)*CONFIGURATIONS + global_conf] = surface_result;
}

// __global__ void compute_surface_layers_minimal(float *max_time, 
//                                                const float *cx, const float *cy, const float *cz, 
//                                                const float *x,  const float *y,  const float *z, const float hor)
// {
//     // const int SURFACE_SIZE       = 5120;
//     const int CONFIGURATION_SIZE = 20;
//     const int CONFIGURATIONS     = 256;
//     const int TIMESTEPS          = 240;

//     const int global_conf  = blockDim.x*blockIdx.x + threadIdx.x; // configuration 
//     const int global_point = blockDim.y*blockIdx.y + threadIdx.y; // point

//     const float cen_x = cx[global_point];
//     const float cen_y = cy[global_point];
//     const float cen_z = cz[global_point];

//     float surf = 0.0, sat_x = 0.0, sat_y = 0.0, sat_z = 0.0, m = 0.0, time = 0.0;
//     for (int timestep = 0; timestep < TIMESTEPS; timestep++) {
//         for (int iorb = 0; iorb < CONFIGURATION_SIZE; iorb++) {
//             sat_x = x[(timestep*CONFIGURATION_SIZE + iorb)*CONFIGURATIONS + global_conf];
//             sat_y = y[(timestep*CONFIGURATION_SIZE + iorb)*CONFIGURATIONS + global_conf];
//             sat_z = z[(timestep*CONFIGURATION_SIZE + iorb)*CONFIGURATIONS + global_conf];
//             surf += (sat_x*cen_x + sat_y*cen_y + sat_z*cen_z > hor);
//         }
//         m    = m*(1.0 - surf) + fmaxf(time, m)*surf;
//         time = (time + 30.0)*(1.0 - surf); 
        

//         // m    = (surf == 0.0) ? fmaxf(m, time) :       m;
//         // time = (surf == 0.0) ? 0.0            : (time + 30.0);
//     }

//     max_time[global_point*CONFIGURATIONS + global_conf] = fmaxf(m, time);
// }

__global__ void reduce_time(float *surface, float *max_time)
{
    const int SURFACE_SIZE       = 5120;
    const int CONFIGURATIONS     = 256;
    const int TIMESTEPS          = 240;

    const int global_conf  = blockDim.x*blockIdx.x + threadIdx.x; // configuration 
    const int global_point = blockDim.y*blockIdx.y + threadIdx.y; // point

    float surf = 0.0; 
    float time = 0.0, m = 0.0; 
    for (int timestep = 1; timestep < TIMESTEPS; timestep++) {
        surf = surface[(timestep*SURFACE_SIZE + global_point)*CONFIGURATIONS + global_conf];
        m    = (surf == 0.0) ? fmaxf(m, time) :       m;
        time = (surf == 0.0) ? 0.0            : (time + 30.0);  
    }
    m = fmaxf(m, time);
    max_time[global_point*CONFIGURATIONS + global_conf] = m;
}

__global__ void reduce_points(float *max_time)
{
    const int SURFACE_SIZE   = 5120;
    const int CONFIGURATIONS = 256;

    const int global_conf  = blockDim.x*blockIdx.x + threadIdx.x; // configuration 

    float max = 0.0; 
    for (int point = 0; point < SURFACE_SIZE; point++)
        max = fmaxf(max, max_time[point*CONFIGURATIONS + global_conf]);
    max_time[global_conf] = max;
}

// // ЛЕВАЯ ГРАНИЦА БЕЗ УГЛОВЫХ ТОЧЕК, ПОДСЧЕТ КОЛИЧЕСТВА ЖИВЫХ СОСЕДЕЙ
// u[0, j] :=   u[1, j]   + u[1, j+1] + u[1, j-1] // ВНУТРИ ОБЛАСТИ
//            + u[0, j+1] + u[0, j-1]           // НА ГРАНИЦЕ
//            + u[N, j]   + u[N, j-1] + u[N, j+1] // БЕРУТСЯ С ПРАВОЙ ГРАНИЦЫ

// // ЛЕВЫЙ ВЕРХНИЙ УГОЛ, ПОДСЧЕТ КОЛИЧЕСТВА ЖИВЫХ СОСЕДЕЙ
// u[0, N] :=   u[1, N-1]                       // ВНУТРИ ОБЛАСТИ
//            + u[1, N]   + u[0, N-1]           // НА ГРАНИЦЕ
//            + u[N, N] +  // БЕРУТСЯ С ПРАВОЙ ГРАНИЦЫ

// max_time = max_time*(1 - surf) + max(time, max_time)*surf;
// time     = (time + settings.deltaT)*(1 - surf);  

// for (size_t j = 0; j < surface.size(); ++j)
//     max_time[j] =                 max_time[j]*(1 - surface[j]) + max(time[j], max_time[j])*surface[j];
// for (size_t j = 0; j < surface.size(); ++j)
//     time[j]     = (time[j] + settings.deltaT)*(1 - surface[j]) +                         0*surface[j]; 

float loc_horizon(const float H, const float alpha)
{
    const float delta = 10.0 * M_PI / 180.0; // требуемое возвышение спутника над горизонтом
    const float alpha_star = asin(cos(delta) / H);

    return (alpha < alpha_star) ? cos(asin(H*sin(alpha)) - alpha) : sin(delta + alpha_star);
    // return sin(delta + alpha_star);
}

float CUDA_Surface::compute_surface(Points1D &gpu_surface, Points1D &gpu_max_time, 
                                    const Points3D &gpu_centroids, const Points3D &gpu_pos)
{
    const int SURFACE_SIZE       = 5120;
    const int CONFIGURATION_SIZE = 20;
    const int CONFIGURATIONS     = 256;
    const int TIMESTEPS = 240;

    dim3    dim_grid_surf(1, SURFACE_SIZE, TIMESTEPS),    dim_block_surf(CONFIGURATIONS, 1, 1);
    dim3 dim_grid_maxtime(1, SURFACE_SIZE, 1),         dim_block_maxtime(CONFIGURATIONS, 1, 1);
    dim3  dim_grid_points(1, 1,            1),          dim_block_points(CONFIGURATIONS, 1, 1);
    // dim3 dim_grid_surf_min(1, SURFACE_SIZE, 1), dim_block_surf_min(CONFIGURATIONS, 1, 1);
    // dim3 dim_grid(CONFIGURATIONS, 1, 1), dim_block_surf(1, SURFACE_SIZE, TIMESTEPS);

    // dim_block.x = CONFIGURATIONS;
    // dim_block.y = 1;
    // dim_block.z = 1;

    // dim_grid.x = 1;
    // dim_grid.y = SURFACE_SIZE;
    // dim_grid.z = TIMESTEPS;

    const float H = (6371.0 + 1500.0) / 6371.0;
    const float alpha = (120.0 * M_PI / 180.0) / 2.0;
    const float hor = loc_horizon(H, alpha);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    compute_surface_layers<<<dim_grid_surf, dim_block_surf>>>(gpu_surface.s, 
                                                              gpu_centroids.x, gpu_centroids.y, gpu_centroids.z,
                                                              gpu_pos.x,       gpu_pos.y,       gpu_pos.z, hor);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float gpu_time = 0.0; 
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("surf = %f ms\n", gpu_time);

    hipEventRecord(start, 0);
    reduce_time<<<dim_grid_maxtime, dim_block_maxtime>>>(gpu_surface.s, gpu_max_time.s);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&gpu_time, start, stop);
    printf("time reduce = %f ms\n", gpu_time);
    
    hipEventRecord(start, 0);
    reduce_points<<<dim_grid_points, dim_block_points>>>(gpu_max_time.s);
    // gpuErrchk( hipPeekAtLastError() );
    // gpuErrchk( hipDeviceSynchronize() );    

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&gpu_time, start, stop);
    printf("point reduce = %f ms\n", gpu_time);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    return gpu_time;          
} 

    // compute_surface_layers<<<dim_grid_surf, dim_block_surf>>>(gpu_surface.s, gpu_centroids.x, gpu_pos.x, hor);
    
// compute_surface_layers_minimal<<<dim_grid_surf_min, dim_block_surf_min>>>(gpu_max_time.s, 
//                                                                           gpu_centroids.x, gpu_centroids.y, gpu_centroids.z,
//                                                                           gpu_pos.x,       gpu_pos.y,       gpu_pos.z, hor);
