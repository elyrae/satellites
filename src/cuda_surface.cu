#include "hip/hip_runtime.h"
#include "cuda_surface.cuh"

#include <cstdio>
#include <hip/hip_runtime.h>

void CUDA_Surface::Points3D::allocate()
{
    hipMalloc((void**) &x, size*sizeof(float));
    hipMalloc((void**) &y, size*sizeof(float));
    hipMalloc((void**) &z, size*sizeof(float));
}

void CUDA_Surface::Points3D::allocate(const int _size)
{
    size = _size;
    hipMalloc((void**) &x, size*sizeof(float));
    hipMalloc((void**) &y, size*sizeof(float));
    hipMalloc((void**) &z, size*sizeof(float));
}

void CUDA_Surface::Points3D::free()
{
    hipFree(x);
    hipFree(y);
    hipFree(z);
}

void CUDA_Surface::Points3D::load_from(const CUDA_Surface::Points3D &cpu) const
{
    hipMemcpy(x, cpu.x, size*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(y, cpu.y, size*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(z, cpu.z, size*sizeof(float), hipMemcpyHostToDevice);
}

void CUDA_Surface::Points3D::save_to(const CUDA_Surface::Points3D &cpu) const
{
    hipMemcpy(cpu.x, x, size*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(cpu.y, y, size*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(cpu.z, z, size*sizeof(float), hipMemcpyDeviceToHost);
}

void CUDA_Surface::Surf::allocate()
{
    hipMalloc((void**) &s, size*sizeof(float));
    hipMemset(s, 0, size);
}

void CUDA_Surface::Surf::allocate(const int _size)
{
    size = _size;
    hipMalloc((void**) &s, size*sizeof(float));
    hipMemset(s, 0, size);
}

void CUDA_Surface::Surf::free()
{
    hipFree(s);
}

void CUDA_Surface::Surf::load_from(const CUDA_Surface::Surf &cpu) const
{
    hipMemcpy(s, cpu.s, size*sizeof(float), hipMemcpyHostToDevice);
}

void CUDA_Surface::Surf::save_to(const CUDA_Surface::Surf &cpu) const
{
    hipMemcpy(cpu.s, s, size*sizeof(float), hipMemcpyDeviceToHost);
}


// __global__ void compute_surface_layer(float *surface, float *cx, float *cy, float *cz, float *x, float *y, float *z, float *h)
// {
//     const int global_time  = blockDim.x*blockIdx.x + threadIdx.x; // time
//     const int global_conf  = blockDim.y*blockIdx.y + threadIdx.y; // configuration 
//     const int global_point = blockDim.z*blockIdx.z + threadIdx.z; // point
    
//     const float sat_h = (6371.0 + 1500.0) / 6371.0;
//     float surface_result = 0.0, sat_x = 0.0, sat_y = 0.0, sat_z = 0.0; 
//     for (int iorb = 0; iorb < CONFIGURATION_SIZE; iorb++)
//     {
//         sat_x = x[(global_time*CONFIGURATIONS + global_conf)*CONFIGURATION_SIZE + iorb];
//         sat_y = y[(global_time*CONFIGURATIONS + global_conf)*CONFIGURATION_SIZE + iorb];
//         sat_z = z[(global_time*CONFIGURATIONS + global_conf)*CONFIGURATION_SIZE + iorb];
//         // sat_h = h[(global_time*CONFIGURATIONS + global_conf)*CONFIGURATION_SIZE + iorb]    
//         surface_result |= (sat_x*cx[global_point] + sat_y*cy[global_point] + sat_z*cz[global_point] > sat_h);
//     } 
//     surface[(global_time*SURFACE_SIZE + global_point)*CONFIGURATIONS + global_conf] = surface_result;
// }