#include "hip/hip_runtime.h"
#include "cuda_surface.cuh"

#include <cstdio>
#include <hip/hip_runtime.h>

void CUDA_Surface::Points3D::allocate()
{
    hipMalloc((void**) &x, size*sizeof(float));
    hipMalloc((void**) &y, size*sizeof(float));
    hipMalloc((void**) &z, size*sizeof(float));
}

void CUDA_Surface::Points3D::allocate(const int _size)
{
    size = _size;
    hipMalloc((void**) &x, size*sizeof(float));
    hipMalloc((void**) &y, size*sizeof(float));
    hipMalloc((void**) &z, size*sizeof(float));
}

void CUDA_Surface::Points3D::free()
{
    hipFree(x);
    hipFree(y);
    hipFree(z);
}

void CUDA_Surface::Points3D::load_from(const CUDA_Surface::Points3D &cpu) const
{
    hipMemcpy(x, cpu.x, size*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(y, cpu.y, size*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(z, cpu.z, size*sizeof(float), hipMemcpyHostToDevice);
}

void CUDA_Surface::Points3D::save_to(const CUDA_Surface::Points3D &cpu) const
{
    hipMemcpy(cpu.x, x, size*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(cpu.y, y, size*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(cpu.z, z, size*sizeof(float), hipMemcpyDeviceToHost);
}

void CUDA_Surface::Points1D::allocate()
{
    hipMalloc((void**) &s, size*sizeof(float));
    hipMemset(s, 0, size);
}

void CUDA_Surface::Points1D::allocate(const int _size)
{
    size = _size;
    hipMalloc((void**) &s, size*sizeof(float));
    hipMemset(s, 0, size);
}

void CUDA_Surface::Points1D::free()
{
    hipFree(s);
}

void CUDA_Surface::Points1D::load_from(const CUDA_Surface::Points1D &cpu) const
{
    hipMemcpy(s, cpu.s, size*sizeof(float), hipMemcpyHostToDevice);
}

void CUDA_Surface::Points1D::save_to(const CUDA_Surface::Points1D &cpu) const
{
    hipMemcpy(cpu.s, s, size*sizeof(float), hipMemcpyDeviceToHost);
}


__global__ void compute_surface_layer(float *surface, const float *cx, const float *cy, const float *cz, 
                                                      const float *x, const float *y, const float *z, const float hor)
{
    const int SURFACE_SIZE       = 5120;
    const int CONFIGURATION_SIZE = 20;
    const int CONFIGURATIONS     = 100;
    const int TIMESTEPS          = 240;

    const int global_conf  = blockDim.x*blockIdx.x + threadIdx.x; // configuration 
    const int global_point = blockDim.y*blockIdx.y + threadIdx.y; // point
    const int global_time  = blockDim.z*blockIdx.z + threadIdx.z; // time

    float surface_result = 0.0, sat_x = 0.0, sat_y = 0.0, sat_z = 0.0; 
    for (int iorb = 0; iorb < CONFIGURATION_SIZE; iorb++)
    {
        sat_x = x[(global_time*CONFIGURATIONS + global_conf)*CONFIGURATION_SIZE + iorb];
        sat_y = y[(global_time*CONFIGURATIONS + global_conf)*CONFIGURATION_SIZE + iorb];
        sat_z = z[(global_time*CONFIGURATIONS + global_conf)*CONFIGURATION_SIZE + iorb];
        // sat_h = h[(global_time*CONFIGURATIONS + global_conf)*CONFIGURATION_SIZE + iorb]    
        surface_result |= (sat_x*cx[global_point] + sat_y*cy[global_point] + sat_z*cz[global_point] > sat_h);
    } 
    surface[(global_time*SURFACE_SIZE + global_point)*CONFIGURATIONS + global_conf] = surface_result;
}